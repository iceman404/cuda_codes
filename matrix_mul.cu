#include <iostream>
#include <hip/hip_runtime.h>

#define N 10

// CUDA kernel for matrix multiplication
__global__ void matrixMultiplication(int* a, int* b, int* c)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        int sum = 0;
        for (int i = 0; i < N; ++i) {
            sum += a[row * N + i] * b[i * N + col];
        }
        c[row * N + col] = sum;
    }
}

int main()
{
    int a[N][N], b[N][N], c[N][N]; // Input and output matrices

    // Initialize input matrices
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            a[i][j] = i + j;
            b[i][j] = i - j;
        }
    }

    int* dev_a, * dev_b, * dev_c; // Device copies of input and output matrices

    // Allocate memory on the device
    hipMalloc((void**)&dev_a, N * N * sizeof(int));
    hipMalloc((void**)&dev_b, N * N * sizeof(int));
    hipMalloc((void**)&dev_c, N * N * sizeof(int));

    // Copy input matrices from host to device
    hipMemcpy(dev_a, a, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * N * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block dimensions for CUDA kernel
    dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(1, 1);

    // Launch the CUDA kernel
    matrixMultiplication<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_c);

    // Copy the result back from the device to the host
    hipMemcpy(c, dev_c, N * N * sizeof(int), hipMemcpyDeviceToHost);

    // Print the output matrix
    std::cout << "Matrix C (Result):" << std::endl;
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            std::cout << c[i][j] << " ";
        }
        std::cout << std::endl;
    }

    // Free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
